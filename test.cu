#include "hip/hip_runtime.h"
#include <vtkm/worklet/DispatcherMapField.h>
#include <vtkm/cont/testing/Testing.h>
#include "KDTree/KdTree.h"

#include <iostream>
#include <chrono>

using namespace std;

#define N_DIMS 2 

namespace
{

using Algorithm = vtkm::cont::Algorithm;

// template< typename DeviceAdapter >
// inline void checkDevice( DeviceAdapter)
// {
//     // using DeviceAdapterTraits = vtkm::cont::DeviceAdapterTraits<DeviceAdapter>;
//     // std::cout << "vtkm is using " << DeviceAdapterTraits::GetName() << std::endl;
// }


// template <typename CoordiVecT, typename CoordiPortalT, typename CoordiT>
// VTKM_EXEC_CONT vtkm::Id NNSVerify3D(CoordiVecT qc, CoordiPortalT coordiPortal, CoordiT& dis)

////brute force method /////
template <typename CoordiVecT, typename CoordiPortalT, typename CoordiT >
VTKM_EXEC_CONT vtkm::Id NNSVerify(CoordiVecT qc, CoordiPortalT coordiPortal, CoordiT& dis)
{
    dis = std::numeric_limits<CoordiT>::max();
    vtkm::Id nnpIdx = 0;

    for (vtkm::Int32 i = 0; i < coordiPortal.GetNumberOfValues(); i++)
    {
        CoordiT _dis = vtkm::Magnitude( coordiPortal.Get( i ) - qc );

        if (_dis < dis)
        {
            dis = _dis;
            nnpIdx = i;
        }
    }

    return nnpIdx;
}

class NearestNeighborSearchBruteForceWorklet : public vtkm::worklet::WorkletMapField
{
public:
    using ControlSignature = void(FieldIn qcIn,
                                  WholeArrayIn treeCoordiIn,
                                  FieldOut nnIdOut,
                                  FieldOut nnDisOut);
    using ExecutionSignature = void(_1, _2, _3, _4);

    VTKM_CONT
    NearestNeighborSearchBruteForceWorklet() {}

    template <typename CoordiVecType, typename CoordiPortalType, typename IdType, typename CoordiType>
    VTKM_EXEC void operator()(const CoordiVecType& qc,
                              const CoordiPortalType& coordiPortal,
                              IdType& nnId,
                              CoordiType& nnDis) const
    {
        nnDis = std::numeric_limits<CoordiType>::max();
        nnId = NNSVerify(qc, coordiPortal, nnDis);
    }
};

void TestKdTreeBuildNNS( vtkm::cont::DeviceAdapterId deviceId )
{
    vtkm::Int32 nTrainingPoints = 300000;
    vtkm::Int32 nTestingPoint   = 300000;

    std::vector<vtkm::Vec< vtkm::Float32, N_DIMS > > coordi;

    ///// randomly generate training points/////
    std::default_random_engine dre;
    std::uniform_real_distribution<vtkm::Float32> dr(0.0f, 10000.0f);

    for (vtkm::Int32 i = 0; i < nTrainingPoints; i++)
    {
        vtkm::Vec< vtkm::Float32, N_DIMS > tp;
        for( int d = 0; d < N_DIMS; ++d )
        {
            tp[ d ] = dr( dre );
        }
        coordi.push_back( tp );
    }

    ///// preprare data to build  kd tree /////
    auto coordi_Handle = vtkm::cont::make_ArrayHandle(coordi);

    vtkm::worklet::KdTree< N_DIMS > kdtree;

    auto t1 = std::chrono::high_resolution_clock::now();

    kdtree.Build( coordi_Handle, deviceId );

    auto t2 = std::chrono::high_resolution_clock::now();
    std::cout << "building took "
              << std::chrono::duration_cast<std::chrono::milliseconds>(t2-t1).count()
              << " milliseconds\n";

    //Nearest Neighbor worklet Testing
    /// randomly generate testing points /////
    std::vector< vtkm::Vec< vtkm::Float32, N_DIMS > > qcVec;
    for (vtkm::Int32 i = 0; i < nTestingPoint; i++)
    {
        vtkm::Vec< vtkm::Float32, N_DIMS > tp;
        for( int d = 0; d < N_DIMS; ++d )
        {
            tp[ d ] = dr( dre );
        }
        qcVec.push_back( tp );
    }

    ///// preprare testing data /////
    auto qc_Handle = vtkm::cont::make_ArrayHandle(qcVec);
    std::vector< vtkm::Float32 > distances( qcVec.size(), std::numeric_limits< float >::max() );

    vtkm::cont::ArrayHandle<vtkm::Id> nnId_Handle;
    auto nnDis_Handle = vtkm::cont::make_ArrayHandle( distances );

    t1 = std::chrono::high_resolution_clock::now();

    // checkDevice( deviceId );

    kdtree.Run(
        coordi_Handle, qc_Handle, nnId_Handle, nnDis_Handle, deviceId );

    t2 = std::chrono::high_resolution_clock::now();
    std::cout << "searching took "
              << std::chrono::duration_cast<std::chrono::milliseconds>(t2-t1).count()
              << " milliseconds\n";

    vtkm::cont::ArrayHandle<vtkm::Id> bfnnId_Handle;
    
    std::vector< vtkm::Float32 > distancesBF( qcVec.size(), std::numeric_limits< float >::max() );
    auto bfnnDis_Handle = vtkm::cont::make_ArrayHandle( distancesBF );

    NearestNeighborSearchBruteForceWorklet nnsbf3dWorklet;

    vtkm::worklet::DispatcherMapField< NearestNeighborSearchBruteForceWorklet > nnsbfDispatcher(
        nnsbf3dWorklet);

    t1 = std::chrono::high_resolution_clock::now();
    
    nnsbfDispatcher.Invoke(
        qc_Handle, vtkm::cont::make_ArrayHandle(coordi), bfnnId_Handle, bfnnDis_Handle );
    
    t2 = std::chrono::high_resolution_clock::now();
    std::cout << "brute force took "
              << std::chrono::duration_cast<std::chrono::milliseconds>(t2-t1).count()
              << " milliseconds\n";

    // ///// verfity search result /////
    bool passTest = true;
    for (vtkm::Int32 i = 0; i < nTestingPoint; i++)
    {
        vtkm::Id workletIdx   = nnId_Handle.GetPortalControl().Get(i);
        vtkm::Id bfworkletIdx = bfnnId_Handle.GetPortalControl().Get(i);

        auto p1 = coordi[ workletIdx   ];
        auto p2 = coordi[ bfworkletIdx ];
        auto p3 = qcVec[ i ];
   
        auto d1 = vtkm::Magnitude( p3 - p1 );
        auto d2 = vtkm::Magnitude( p3 - p2 );

        if ( workletIdx != bfworkletIdx && d1 != d2 )
        {
            std::cout << workletIdx << "!=" << bfworkletIdx << " at " << i << std::endl;
            std::cout << vtkm::Magnitude( p3 - p1 ) << " vs " << vtkm::Magnitude( p3 - p2 ) << std::endl;
            passTest = false;
        }
    }

    if( passTest )
    {
        std::cout << "Passed Test\n";
    }

    VTKM_TEST_ASSERT( passTest, "Kd tree NN search result incorrect.");
}

} // anonymous namespace

int main(int argc, char* argv[])
{
    vtkm::cont::testing::Testing::RunOnDevice( TestKdTreeBuildNNS, argc, argv );
}
